#include <hip/hip_runtime.h>

__global__ void addKernel(int *a, int *b, int *c) {
    *c = *a + *b;
}

extern "C" {
    int addNumbers(int a, int b) {
        int *d_a, *d_b, *d_c;
        int result;
        
        hipMalloc((void**)&d_a, sizeof(int));
        hipMalloc((void**)&d_b, sizeof(int));
        hipMalloc((void**)&d_c, sizeof(int));
        
        hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);
        
        addKernel<<<1,1>>>(d_a, d_b, d_c);
        
        hipMemcpy(&result, d_c, sizeof(int), hipMemcpyDeviceToHost);
        
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
        
        return result;
    }
}